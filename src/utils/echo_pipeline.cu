#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <cmath>
#include <complex>
#include <iostream>

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ << ": " \
                  << hipGetErrorString(err) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}
#define CHECK_CUFFT(call) { \
    hipfftResult err = call; \
    if (err != HIPFFT_SUCCESS) { \
        std::cerr << "CUFFT error in " << __FILE__ << " at line " << __LINE__ << ": " \
                  << err << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

// Kernel: Build Gaussian Packet F_j(k)

__global__ void gaussian_wavepacket(hipfftDoubleComplex* f, const double* k, double k0, double x0, double sigma, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        double arg = -((k[i] - k0) * (k[i] - k0)) / (4.0 * sigma * sigma);
        double phase = k[i] * x0;
        double pref = pow(1.0 / (2.0 * M_PI * sigma * sigma), 0.25);
        double gauss = pref * exp(arg);
        f[i].x = gauss * cos(phase);
        f[i].y = gauss * sin(phase);
    }
}

// Compute Overlap g(k,t)
__global__ void compute_overlap(hipfftDoubleComplex* g, const hipfftDoubleComplex* f, const hipfftDoubleComplex* psi, const double* omega, double t, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        double real_part = f[i].x * psi[i].x + f[i].y * psi[i].y;
        double imag_part = f[i].y * psi[i].x - f[i].x * psi[i].y; // conjugate multiplication
        double phase = -omega[i]*t;
        double norm = rsqrt(4.0 * M_PI * omega[i] + 1e-12);
        double cos_phase = cos(phase);
        double sin_phase = sin(phase);
        g[i].x = norm * (real_part * cos_phase - imag_part * sin_phase);
        g[i].y = norm * (real_part * sin_phase + imag_part * cos_phase);
    }
}

// Reduction compute E_tot(t) = ∫ dx |E(x,t) |^2

double compute_Etot(const hipfftDoubleComplex* E, int N) {
    thrust::device_ptr<const hipfftDoubleComplex> dptr(E);
    auto sq_norm = [=] __device__(hipfftDoubleComplex z) {
        return z.x*z.x + z.y*z.y;
    };
    return thrust::transform_reduce(dptr, dptr+N, sq_norm, 0.0, thrust::plus<double>());
}

// Entry point (called by the bindings)
extern "C"
void run_echo_pipeline(double* k_host, double* omega_host, int N, double k1, double x1, double k2, double x2, double sigma, double t, hipfftDoubleComplex* E_out) {
    double *d_k, *d_omega;
    hipfftDoubleComplex *f1, *f2, *g, *E;
    CHECK_CUDA(hipMalloc(&d_k, N*sizeof(double)));
    CHECK_CUDA(hipMalloc(&d_omega, N*sizeof(double)));
    CHECK_CUDA(hipMalloc(&f1, N*sizeof(hipfftDoubleComplex)));
    CHECK_CUDA(hipMalloc(&f2, N*sizeof(hipfftDoubleComplex)));
    CHECK_CUDA(hipMalloc(&g, N*sizeof(hipfftDoubleComplex)));
    CHECK_CUDA(hipMalloc(&E, N*sizeof(hipfftDoubleComplex)));

    // copy K, omega
    CHECK_CUDA(hipMemcpy(d_k, k_host, N*sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_omega, omega_host, N*sizeof(double), hipMemcpyHostToDevice));
    // Launch Gaussian Builders
    int threads = 256;
    int blocks = (N + threads -1) / threads;
    gaussian_wavepacket<<<blocks, threads>>>(f1, d_k, k1, x1, sigma, N);
    gaussian_wavepacket<<<blocks, threads>>>(f2, d_k, k2, x2, sigma, N);
    // compute g(k, t)
    compute_overlap<<<blocks, threads>>>(g, f1, f2, d_omega, t, N);
    // FFT g(k,t) -> E(x, t)
    hipfftHandle plan;
    CHECK_CUFFT(hipfftPlan1d(&plan, N, HIPFFT_Z2Z, 1));
    CHECK_CUFFT(hipfftExecZ2Z(plan, g, E, HIPFFT_BACKWARD));
    CHECK_CUFFT(hipfftDestroy(plan));
    // copy E(x, t) to host
    CHECK_CUDA(hipMemcpy(E_out, E, N*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost));
    // Compute E_tot
    double E_tot = compute_Etot(E, N);
    // Free
    hipFree(d_k); hipFree(d_omega);
    hipFree(f1); hipFree(f2); hipFree(g); hipFree(E);
}